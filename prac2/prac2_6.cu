#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

__constant__ float a, b, c;

__global__ void aver(float *z, float *res) {
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   float tmp = 0;

   for (int i = 0; i < 100; i++) {
      tmp += a * z[tid] * z[tid] + b * z[tid] + c;
      tid += blockIdx.x * blockDim.x;
   }
   res[threadIdx.x] = tmp/100;
}

int main() {
   int n_thread = 256;
   int N = n_thread * 100;
   float *d_z, *h_res, *d_res;
   float h_a, h_b, h_c;

   h_res = (float *)malloc(sizeof(float) * n_thread);

   checkCudaErrors(hipMalloc((void **)&d_z, sizeof(float)*N));
   checkCudaErrors(hipMalloc((void**)&d_res, sizeof(float)*n_thread));

   h_a = 1.0f;
   h_b = 2.0f;
   h_c = 3.0f;

   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(a), &h_a, sizeof(h_a)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(b), &h_b, sizeof(h_b)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c), &h_c, sizeof(h_c)));

   hiprandGenerator_t gen;
   checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
   checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );

   checkCudaErrors( hiprandGenerateNormal(gen, d_z, N, 0.0f, 1.0f) );

   hipEvent_t start, end;
   float milli;

   hipEventCreate(&start);
   hipEventCreate(&end);

   hipEventRecord(start);
   aver<<<1, n_thread>>>(d_z, d_res);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);

   checkCudaErrors( hipMemcpy(h_res, d_res, sizeof(float)*n_thread,
                   hipMemcpyDeviceToHost) );

   float sum = 0.0f;
   for (int i = 0; i < n_thread; i++) {
      sum += h_res[i];
   }
   printf("average val == a + c: %f == %f + %f\ntime comsumed: %f\n", sum/n_thread, h_a, h_c, milli);

   checkCudaErrors( hiprandDestroyGenerator(gen) );

   free(h_res);
   checkCudaErrors(hipFree(d_z));
   checkCudaErrors(hipFree(d_res));

   hipDeviceReset();

   return 0;
}