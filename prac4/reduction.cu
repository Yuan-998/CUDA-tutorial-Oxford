#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////////////
// CPU routines
////////////////////////////////////////////////////////////////////////////////

void reduction_gold(float* odata, float* idata, const unsigned int len) 
{
  *odata = 0;
  for(int i=0; i<len; i++) *odata += idata[i];
}

////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////

__global__ void reduction(float *g_odata, float *g_idata, int n)
{
    // dynamically allocated shared memory

    extern  __shared__  float temp[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // first, each thread loads data into shared memory

    temp[tid] = g_idata[tid];

    // next, we perform binary tree reduction

    for (int d = blockDim.x>>1; d > 0; d >>= 1) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d+blockIdx.x * blockDim.x)  temp[tid] += temp[tid+d];
    }

    // finally, first thread puts result into global memory

    if (tid==blockIdx.x * blockDim.x) g_odata[tid] = temp[tid];
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, const char** argv) 
{
  int num_elements, num_block, num_threads, mem_size, shared_mem_size;

  float *h_data, *reference, sum;
  float *d_idata, *d_odata;

  // initialise card

  findCudaDevice(argc, argv);

  num_elements = 512;
  num_block = 2;
  num_threads  = num_elements/num_block;
  mem_size     = sizeof(float) * num_elements;

  // allocate host memory to store the input data
  // and initialize to integer values between 0 and 1000

  h_data = (float*) malloc(mem_size);
      
  for(int i = 0; i < num_elements; i++) 
    h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));

  // compute reference solutions

  reference = (float*) malloc(mem_size);
  reduction_gold(&sum, h_data, num_elements);

  // allocate device memory input and output arrays

  checkCudaErrors( hipMalloc((void**)&d_idata, mem_size) );
  checkCudaErrors( hipMalloc((void**)&d_odata, mem_size) );

  // copy host memory to device input array

  checkCudaErrors( hipMemcpy(d_idata, h_data, mem_size,
                              hipMemcpyHostToDevice) );

  // execute the kernel

  shared_mem_size = sizeof(float) * num_elements;
  reduction<<<num_block,num_threads,shared_mem_size>>>(d_odata,d_idata, num_elements);
  getLastCudaError("reduction kernel execution failed");

  // copy result from device to host

  checkCudaErrors( hipMemcpy(h_data, d_odata, mem_size,
                              hipMemcpyDeviceToHost) );

  float sum_gpu = 0.0f;
  for (int i = 0; i < num_block; i++)
  {
    sum_gpu += h_data[i * num_threads];
  }
  
  // check results
  printf("reduction error = %f\n",sum_gpu-sum);

  // cleanup memory

  free(h_data);
  free(reference);
  checkCudaErrors( hipFree(d_idata) );
  checkCudaErrors( hipFree(d_odata) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();
}
